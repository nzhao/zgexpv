#include "hip/hip_runtime.h"
// |w> = |w> + H * |v>;
// H = sum(H_i) + sum(H_ij);
// H_i = h_i * S_(i,xyz) or h_ij * S_(i,xyz) * S_(j,xyz);
// w_med = H_ij * w_med 

#include <iostream>
#include <complex>
#include <cmath>

#include "hip/hip_runtime.h"
#include "hipblas.h"

// global variable used for texture memory optimization;
texture< int2, 1, hipReadModeElementType > texRef;

//======================================================================
// kron_cuda_v1, dev_v -> dev_w_med;
//======================================================================
__global__ void kron_cuda_v1( const size_t m, const size_t s, const size_t n, const hipDoubleComplex *A, size_t mat_i_idx_idx, hipDoubleComplex *x, hipDoubleComplex *w )
{
  hipDoubleComplex res;
  hipDoubleComplex mid;
  int2 a1,a2;  

  size_t k;
  size_t q, idx2;
  
  extern __shared__ hipDoubleComplex x_shd[ ];
  
  k = blockDim.y * ( blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z ) + threadIdx.y;
  
//  idx2 = s * n * ( k / n ) + k % n;
  idx2 = ( s - 1 ) * n * ( k / n ) + k;
  
  // copy x to the shared memory x_shd;
  x_shd[ threadIdx.x + blockDim.x * threadIdx.y ] = x[ idx2 + n * threadIdx.x];
  __syncthreads();
  
  // matrix multiplication using the shared memory;
  //res = hipCmul( x_shd[ blockDim.x * threadIdx.y ], A[ threadIdx.x ] );
  a1 = tex1Dfetch( texRef, ( mat_i_idx_idx + threadIdx.x ) * 2 );
  a2 = tex1Dfetch( texRef, ( mat_i_idx_idx + threadIdx.x ) * 2 + 1 );
  mid.x = __hiloint2double( a1.y, a1.x );
  mid.y = __hiloint2double( a2.y, a2.x );
  
  res = hipCmul( x_shd[ blockDim.x * threadIdx.y ], mid);
  for ( q = 1; q < s; q++ )
  {
    a1 = tex1Dfetch( texRef, ( mat_i_idx_idx + threadIdx.x + s * q ) * 2 );
    a2 = tex1Dfetch( texRef, ( mat_i_idx_idx + threadIdx.x + s * q ) * 2 + 1 );
    mid.x = __hiloint2double( a1.y, a1.x );
    mid.y = __hiloint2double( a2.y, a2.x );
    res = hipCadd( res, hipCmul( x_shd[ q + blockDim.x * threadIdx.y ], mid) );
  }
  
  // copy to the global memory x;
  w[ idx2 + n * threadIdx.x ] = res;
}
//======================================================================

//======================================================================
// kron_cuda_v2, dev_w_med -> dev_w_med;
//======================================================================
__global__ void kron_cuda_v2( const size_t m, const size_t s, const size_t n, const hipDoubleComplex *A, size_t mat_i_idx_idx, hipDoubleComplex *x )
{
  hipDoubleComplex res;
  hipDoubleComplex mid;
  
  int2   a1, a2;  

  size_t k;
  size_t q, idx2;
  
  extern __shared__ hipDoubleComplex x_shd[ ];
//  extern __shared__ hipDoubleComplex y_shd[ ];
  
  k = blockDim.y * ( blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z ) + threadIdx.y;
  
//  idx2 = s * n * ( k / n ) + k % n;
  idx2 = ( s - 1 ) * n * ( k / n ) + k; 
  
  // copy x to the shared memory x_shd;
  x_shd[ threadIdx.x + blockDim.x * threadIdx.y ] = x[ idx2 + n * threadIdx.x];
  __syncthreads();
  
  // matrix multiplication using the shared memory;
  //res = hipCmul( x_shd[ blockDim.x * threadIdx.y ], A[ threadIdx.x ] );
  a1 = tex1Dfetch( texRef, ( mat_i_idx_idx + threadIdx.x ) * 2 );
  a2 = tex1Dfetch( texRef, ( mat_i_idx_idx + threadIdx.x ) * 2 + 1 );
  mid.x = __hiloint2double( a1.y, a1.x );
  mid.y = __hiloint2double( a2.y, a2.x );
  
  res = hipCmul( x_shd[ blockDim.x * threadIdx.y ], mid );
  for ( q = 1; q < s; q++ )
  {
    a1 = tex1Dfetch( texRef, ( mat_i_idx_idx + threadIdx.x + s * q ) * 2 );
    a2 = tex1Dfetch( texRef, ( mat_i_idx_idx + threadIdx.x + s * q ) * 2 + 1 );
    mid.x = __hiloint2double( a1.y, a1.x );
    mid.y = __hiloint2double( a2.y, a2.x );
    res = hipCadd( res, hipCmul( x_shd[ q + blockDim.x * threadIdx.y ], mid ) );
  }
//  y_shd[ threadIdx.x + blockDim.x * threadIdx.y ] = res;
//  __syncthreads();
  
  // copy y_shd to the global memory x;
//  x[ idx2 + n * threadIdx.x ] = y_shd[ threadIdx.x + blockDim.x * threadIdx.y ];
  x[ idx2 + n * threadIdx.x ] = res;
}
//======================================================================

//======================================================================
// kron_cuda_v3, dev_w_med -> dev_w;
//======================================================================
__global__ void kron_cuda_v3( const size_t m, const size_t s, const size_t n, const hipDoubleComplex *A, size_t mat_i_idx_idx, hipDoubleComplex *x, hipDoubleComplex *w, hipDoubleComplex *coeff_lst, size_t nT )
{
  hipDoubleComplex res;
  hipDoubleComplex mid;
  int2 a1,a2;  

  size_t k;
  size_t q, idx2;
  
  extern __shared__ hipDoubleComplex x_shd[ ];
  
  k = blockDim.y * ( blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z ) + threadIdx.y;
  
//  idx2 = s * n * ( k / n ) + k % n;
  idx2 = ( s - 1 ) * n * ( k / n ) + k;
  
  // copy x to the shared memory x_shd;
  x_shd[ threadIdx.x + blockDim.x * threadIdx.y ] = x[ idx2 + n * threadIdx.x];
  __syncthreads();
  
  // matrix multiplication using the shared memory;
  //res = hipCmul( x_shd[ blockDim.x * threadIdx.y ], A[ threadIdx.x ] );
  a1 = tex1Dfetch( texRef, ( mat_i_idx_idx + threadIdx.x ) * 2 );
  a2 = tex1Dfetch( texRef, ( mat_i_idx_idx + threadIdx.x ) * 2 + 1 );
  mid.x = __hiloint2double( a1.y, a1.x );
  mid.y = __hiloint2double( a2.y, a2.x );
  
  res = hipCmul( x_shd[ blockDim.x * threadIdx.y ], mid);
  for ( q = 1; q < s; q++ )
  {
    a1 = tex1Dfetch( texRef, ( mat_i_idx_idx + threadIdx.x + s * q ) * 2 );
    a2 = tex1Dfetch( texRef, ( mat_i_idx_idx + threadIdx.x + s * q ) * 2 + 1 );
    mid.x = __hiloint2double( a1.y, a1.x );
    mid.y = __hiloint2double( a2.y, a2.x );
    res = hipCadd( res, hipCmul( x_shd[ q + blockDim.x * threadIdx.y ], mid) );
  }
  
  // copy to the global memory x;
  res = hipCmul( res, coeff_lst[ nT ] );
  w[ idx2 + n * threadIdx.x ] = hipCadd( w[ idx2 + n * threadIdx.x ], res );
}
//======================================================================

__global__ void vecrzt_kernel( hipDoubleComplex *x )
{
  size_t  idx;
  
  idx = blockDim.x * ( blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z ) + threadIdx.x;
  
  x[ idx ] = make_hipDoubleComplex(0.0,0.0);
}

void hamvec_cuda3( hipblasHandle_t cublas_handle, int nspin, int nTerm, std::complex<double> *coeff_lst_zplx, size_t *nbody_lst, size_t *pos_i_idx, size_t *pos_i_lst, size_t *dim_i_lst, size_t *mat_i_idx, std::complex<double> *dev_mat_i_lst, size_t vlen, size_t *nspin_dim, size_t *nspin_m_lst, size_t *nspin_n_lst, std::complex<double> *dev_v, std::complex<double> *dev_w, std::complex<double> *dev_w_med, std::complex<double> *dev_coeff_lst_zplx, size_t maxThreadsPerBlock, size_t *maxGridSize )
{
/*
  Calculate the action of a Hamiltonian on a state vector.
  The Hamiltonian can be decomposed into many terms, where each term 
  consists of 
  
  input:
    
    cublas_handle,  handle of cublas;
    nspin,          number of bodies;
    nTerm,          number of interactions in the Hamiltonian;
    coeff_lst_zplx, list of the coefficient in each interaction;
    nbody_lst,      list of the number of bodies in each interaction;
    pos_i_idx,      list of the index of the position list of the body 
                    in each interaction;
    pos_i_lst,      list of the positions of bodies in each interaction;
    dim_i_lst,      list of the dimension of operator of each body in 
                    each interaction;
    mat_i_idx,      list of the index of the operator list of the body
                    in each interaction;
    dev_mat_i_lst,  list of the operators of bodies in each interaction; 
    vlen,           dimension of the state vector;
    nspin_dim,      list of the dimension of each body;
    nspin_m_lst,    list of the dimension of first m bodies;
    nspin_n_lst,    list of the dimension of last n bodies;
    dev_v,          input state vector;
    dev_w_med,      intermediate state vector;
    dev_coeff_lst_zplx, the same as "coeff_lst_zplx";
    maxThreadsPerBlock,   max block size for CUDA;
    maxGridSize,    grid size limit for CUDA;
  
  output:
    
    dev_w,          output state vector, after the Hamiltonian acting on
                    the input state;
*/
  
  size_t  nT, nbody, nb;
  size_t  idx, pos_i, dim_i;
  std::complex<double>  coeff;
  
  size_t  m, n;
  
  dim3    grid_dim, block_dim;
  size_t  dimex1, dimex2;
  
  // optimization by texture memory, bind the operator list;
  hipBindTexture( 0, texRef, dev_mat_i_lst );
  
  //====================================================================
  // initialization of w;
  //====================================================================
  
  // set block size, 2D;
  block_dim.z = 1;
  block_dim.y = 1;
  dimex1 = maxThreadsPerBlock;
  dimex2 = vlen;
  while ( (dimex2 % dimex1) != 0 ) dimex1--;
  block_dim.x = dimex1;
  dimex2 /= dimex1;
  if ( dimex2 <= maxGridSize[0] * maxGridSize[1] * maxGridSize[2] )
  {
    dimex1 = dimex2 / ( maxGridSize[0] * maxGridSize[1] );
    if ( ( dimex2 % ( maxGridSize[0] * maxGridSize[1] ) ) > 0 )
      dimex1++;
    while ( (dimex2 % dimex1) != 0 ) dimex1++;
    grid_dim.z = dimex1;
    dimex2 = dimex2 / grid_dim.z;
    dimex1 = dimex2 / maxGridSize[0];
    if ( ( dimex2 % maxGridSize[0] ) > 0)
      dimex1++;
    while ( (dimex2 % dimex1) != 0 ) dimex1++;
    grid_dim.y = dimex1;
    grid_dim.x = dimex2 / grid_dim.y;
  }
  else
  {
    std::cout << "block number exceeds limit." << std::endl;
    return;
  }
  
  // reset the output vector;
  vecrzt_kernel<<< grid_dim, block_dim >>>( (hipDoubleComplex*)dev_w );
  
  //====================================================================
  // action of the Hamiltonian on the input state;
  //====================================================================
  
  // loop over each term of the Hamiltonian;
  for ( nT = 0; nT < nTerm; nT++ )
  {
    coeff = coeff_lst_zplx[ nT ];
    nbody = nbody_lst[ nT ];
    
    if ( abs(coeff) == 0 ) continue;
    
    // loop over each body in each term of the Hamiltonian;
    for ( nb = 0; nb < nbody; nb++ )
    {
      idx = pos_i_idx[ nT ] + nbody - 1 - nb;
      pos_i = pos_i_lst[ idx ];
      dim_i = dim_i_lst[ idx ];
      
      m = nspin_m_lst[ pos_i ];
      n = nspin_n_lst[ pos_i ];
      
      // set block size, 2D;
      block_dim.z = 1;
      block_dim.x = dim_i;
      dimex1 = maxThreadsPerBlock / block_dim.x;
      dimex2 = m * n;
      while ( (dimex2 % dimex1) != 0 ) dimex1--;
      block_dim.y = dimex1;
      // set grid size, 1D-3D;
      dimex2 /= dimex1;// grid size is m * n / block_dim.y;
      if ( dimex2 <= maxGridSize[0] * maxGridSize[1] * maxGridSize[2] )
      {
        dimex1 = dimex2 / ( maxGridSize[0] * maxGridSize[1] );
        if ( ( dimex2 % ( maxGridSize[0] * maxGridSize[1] ) ) > 0 )
          dimex1++;
        while ( (dimex2 % dimex1) != 0 ) dimex1++;
        grid_dim.z = dimex1;
        dimex2 = dimex2 / grid_dim.z;
        dimex1 = dimex2 / maxGridSize[0];
        if ( ( dimex2 % maxGridSize[0] ) > 0)
          dimex1++;
        while ( (dimex2 % dimex1) != 0 ) dimex1++;
        grid_dim.y = dimex1;
        grid_dim.x = dimex2 / grid_dim.y;
      }
      else
      {
        std::cout << "block number exceeds limit." << std::endl;
        return;
      }
      
      if ( nbody == 1 )
        kron_cuda_v3<<< grid_dim, block_dim, block_dim.x * block_dim.y * sizeof(std::complex<double>) >>>( m, dim_i, n, (hipDoubleComplex*)dev_mat_i_lst, mat_i_idx[idx], (hipDoubleComplex*)dev_v, (hipDoubleComplex*)dev_w, (hipDoubleComplex*)dev_coeff_lst_zplx, nT );
      else
      {
        if ( nb == 0 )
          kron_cuda_v1<<< grid_dim, block_dim, block_dim.x * block_dim.y * sizeof(std::complex<double>) >>>( m, dim_i, n, (hipDoubleComplex*)dev_mat_i_lst, mat_i_idx[idx], (hipDoubleComplex*)dev_v, (hipDoubleComplex*)dev_w_med );
        else if ( nb == nbody - 1 )
          kron_cuda_v3<<< grid_dim, block_dim, block_dim.x * block_dim.y * sizeof(std::complex<double>) >>>( m, dim_i, n, (hipDoubleComplex*)dev_mat_i_lst, mat_i_idx[idx], (hipDoubleComplex*)dev_w_med, (hipDoubleComplex*)dev_w, (hipDoubleComplex*)dev_coeff_lst_zplx, nT );
        else
          kron_cuda_v2<<< grid_dim, block_dim, block_dim.x * block_dim.y * sizeof(std::complex<double>) >>>( m, dim_i, n, (hipDoubleComplex*)dev_mat_i_lst, mat_i_idx[idx], (hipDoubleComplex*)dev_w_med );
      }
      
    }
    
  }
  
  // optimization by texture memory, release the bind;
  hipUnbindTexture( texRef );
}

//======================================================================
// interface for FORTRAN;
//======================================================================

// declaration of function handle;
#define HAMVEC_CUDA3       hamvec_cuda3_
#define HAMVEC_CUDA3_INIT  hamvec_cuda3_init_
#define HAMVEC_CUDA3_TERM  hamvec_cuda3_term_

// delcaration of function;
#if defined(__cplusplus)
extern "C" {
#endif /* __cplusplus */

void HAMVEC_CUDA3( size_t *cublas_handle_ptr, int *nspin_ptr, int *nTerm_ptr, std::complex<double> *coeff_lst_zplx_ptr, size_t *nbody_lst_ptr, size_t *pos_i_idx_ptr, size_t *pos_i_lst_ptr, size_t *dim_i_lst_ptr, size_t *mat_i_idx_ptr, size_t *dev_mat_i_lst_ptr, size_t *ham_dim_ptr, size_t *nspin_dim_ptr, size_t *nspin_m_lst_ptr, size_t *nspin_n_lst_ptr, size_t *dev_v_ptr, size_t *dev_w_ptr, size_t *dev_w_med_ptr, size_t *dev_coeff_lst_zplx_ptr, size_t *maxThreadsPerBlock_ptr, size_t *maxGridSize_ptr );

void HAMVEC_CUDA3_INIT( size_t *cublas_handle_ptr );

void HAMVEC_CUDA3_TERM( size_t *cublas_handle_ptr );

#if defined(__cplusplus)
}
#endif /* __cplusplus */

// interface of function;
void HAMVEC_CUDA3( size_t *cublas_handle_ptr, int *nspin_ptr, int *nTerm_ptr, std::complex<double> *coeff_lst_zplx_ptr, size_t *nbody_lst_ptr, size_t *pos_i_idx_ptr, size_t *pos_i_lst_ptr, size_t *dim_i_lst_ptr, size_t *mat_i_idx_ptr, size_t *dev_mat_i_lst_ptr, size_t *ham_dim_ptr, size_t *nspin_dim_ptr, size_t *nspin_m_lst_ptr, size_t *nspin_n_lst_ptr, size_t *dev_v_ptr, size_t *dev_w_ptr, size_t *dev_w_med_ptr, size_t *dev_coeff_lst_zplx_ptr, size_t *maxThreadsPerBlock_ptr, size_t *maxGridSize_ptr )
{
  hipblasHandle_t        cublas_handle   = (hipblasHandle_t)*cublas_handle_ptr;
  int                   nspin           = *nspin_ptr;
  int                   nTerm           = *nTerm_ptr;
  std::complex<double>  *coeff_lst_zplx = coeff_lst_zplx_ptr;
  size_t                *nbody_lst      = nbody_lst_ptr;
  size_t                *pos_i_idx      = pos_i_idx_ptr;
  size_t                *pos_i_lst      = pos_i_lst_ptr;
  size_t                *dim_i_lst      = dim_i_lst_ptr;
  size_t                *mat_i_idx      = mat_i_idx_ptr;
  std::complex<double>  *dev_mat_i_lst  = (std::complex<double>*)(*dev_mat_i_lst_ptr);
  size_t                vlen            = *ham_dim_ptr;
  size_t                *nspin_dim      = nspin_dim_ptr;
  size_t                *nspin_m_lst    = nspin_m_lst_ptr;
  size_t                *nspin_n_lst    = nspin_n_lst_ptr;
  std::complex<double>  *dev_v          = (std::complex<double>*)(*dev_v_ptr);
  std::complex<double>  *dev_w          = (std::complex<double>*)(*dev_w_ptr);
  std::complex<double>  *dev_w_med      = (std::complex<double>*)(*dev_w_med_ptr);
  std::complex<double>  *dev_coeff_lst_zplx = (std::complex<double>*)(*dev_coeff_lst_zplx_ptr);
  size_t            maxThreadsPerBlock  = *maxThreadsPerBlock_ptr;
  size_t                *maxGridSize    = maxGridSize_ptr;
  
  hamvec_cuda3( cublas_handle, nspin, nTerm, coeff_lst_zplx, nbody_lst, pos_i_idx, pos_i_lst, dim_i_lst, mat_i_idx, dev_mat_i_lst, vlen, nspin_dim, nspin_m_lst, nspin_n_lst, dev_v, dev_w, dev_w_med, dev_coeff_lst_zplx, maxThreadsPerBlock, maxGridSize );
}

void HAMVEC_CUDA3_INIT( size_t *cublas_handle_ptr )
{
  // initialization of cublas handle for FORTRAN;
  hipblasHandle_t cublas_handle;
  hipblasCreate( &cublas_handle );
  *cublas_handle_ptr = (size_t)cublas_handle;
}

void HAMVEC_CUDA3_TERM( size_t *cublas_handle_ptr )
{
  // termination of cublas handle for FORTRAN;
  hipblasHandle_t cublas_handle;
  cublas_handle = (hipblasHandle_t)*cublas_handle_ptr;
  hipblasDestroy( cublas_handle );
}

