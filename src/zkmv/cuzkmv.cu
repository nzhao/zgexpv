/*
  Calculation of the matrix-vector multiplication exploited by CUDA, where the matrix can be decomposed into sum of the kronecker multiplication.
  
  cuzkmv( nspin, nTerm, coeff_lst, nbody_lst, pos_i_idx, pos_i_lst, dim_i_lst, mat_i_idx, mat_i_lst, vlen, nspin_dim, v, w );
  
  input:
    
    nspin,                  integer, 
                            number of total spins;
    nTerm,                  integer, 
                            number of Hamiltonian terms;
    coeff_lst[ nTerm ],     double, 
                            coefficient list in each term;
    nbody_lst[ nTerm ],     size_t, 
                            spin number in each term;
    pos_idx,                size_t, 
                            total length of spin position list, the same as
                            sum of nbody_lst;
    pos_i_idx[ nTerm ],     size_t, 
                            offset of the spin position list;
    pos_i_lst[ pos_idx ],   size_t, 
                            spin position list;
    dim_i_lst[ pos_idx ],   size_t,
                            spin dimension list;
    mat_idx,                size_t, 
                            total length of the spin operator list;
    mat_i_idx[ pos_idx ],   size_t, 
                            offset of the spin operator list;
    mat_i_lst[ mat_idx ],          complex<double>, 
                            spin operator list;
    vlen,                   size_t, 
                            length of the state vector;
    v[ vlen ],              complex<double>, 
                            input state vector;
    nspin_dim[ nspin ],     size_t, 
                            list of spin dimension;
    
  output:
    
    w[ vlen ],              complex<double>, 
                            output state vector;
*/

#include <iostream>
#include <complex>
#include <cmath>

#include "hip/hip_runtime.h"
#include "hipblas.h"

void hamvec_cuda3( hipblasHandle_t cublas_handle, int nspin, int nTerm, std::complex<double> *coeff_lst_zplx, size_t *nbody_lst, size_t *pos_i_idx, size_t *pos_i_lst, size_t *dim_i_lst, size_t *mat_i_idx, std::complex<double> *dev_mat_i_lst, size_t vlen, size_t *nspin_dim, size_t *nspin_m_lst, size_t *nspin_n_lst, std::complex<double> *dev_v, std::complex<double> *dev_w, std::complex<double> *dev_w_med, std::complex<double> *dev_coeff_lst_zplx, size_t maxThreadsPerBlock, size_t *maxGridSize );

void cuzkmv( int nspin, int nTerm, double *coeff_lst, size_t *nbody_lst, size_t *pos_i_idx, size_t *pos_i_lst, size_t *dim_i_lst, size_t *mat_i_idx, std::complex<double> *mat_i_lst, size_t vlen, size_t *nspin_dim, std::complex<double> *v, size_t pos_idx, size_t mat_idx, std::complex<double> *w )
{
  
  // coefficients for matrix;
  std::complex<double>  *coeff_lst_zplx;
  size_t                *nspin_m_lst, *nspin_n_lst;
  
  // CUDA;
  // define grid size;
  size_t maxThreadsPerBlock = 256;
  size_t maxGridSize[3]     = {2147483647,65535,65535};
  
  // variables on device;
  hipError_t           cuda_status;
  hipblasHandle_t        cublas_handle;
  std::complex<double>  *dev_mat_i_lst;
  std::complex<double>  *dev_v, *dev_w, *dev_w_med;
  std::complex<double>  *dev_coeff_lst_zplx;
  
  size_t                i, j, k;
  
  coeff_lst_zplx = new std::complex<double> [ nTerm ];
  for ( i = 0; i < nTerm; i++ )
    coeff_lst_zplx[i] = std::complex<double> ( coeff_lst[i], 0.0  );// h;
//    coeff_lst_zplx[i] = std::complex<double> ( 0.0, -coeff_lst[i] );// -i*h;
  
  nspin_m_lst = new size_t [ nspin ];
  nspin_n_lst = new size_t [ nspin ];
  for ( i = 0; i < nspin; i++ )
  {
    k = 1;
    for ( j = 0; j < i; j++ )
      k = k * nspin_dim[ j ];
    nspin_m_lst[ i ] = k;
    k = 1;
    for ( j = i + 1; j < nspin; j++ )
      k = k * nspin_dim[ j ];
    nspin_n_lst[ i ] = k;
  }
  
  // malloc on device;
  cuda_status = hipMalloc((void**)&dev_mat_i_lst, mat_idx * sizeof( mat_i_lst[0] ) );
  if (cuda_status != hipSuccess)
    std::cout << "Device malloc failed: dev_mat_i_lst" << std::endl;
  cuda_status = hipMalloc((void**)&dev_v, vlen * sizeof( v[0] ) );
  if (cuda_status != hipSuccess)
    std::cout << "Device malloc failed: dev_v" << std::endl;
  cuda_status = hipMalloc((void**)&dev_w, vlen * sizeof( w[0] ) );
  if (cuda_status != hipSuccess)
    std::cout << "Device malloc failed: dev_w" << std::endl;
  cuda_status = hipMalloc((void**)&dev_w_med, vlen * sizeof( v[0] ) );
  if (cuda_status != hipSuccess)
    std::cout << "Device malloc failed: dev_w_med" << std::endl;
  cuda_status = hipMalloc((void**)&dev_coeff_lst_zplx, nTerm * sizeof( coeff_lst_zplx[0] ) );
  if (cuda_status != hipSuccess)
    std::cout << "Device malloc failed: dev_coeff_lst_zplx" << std::endl;
  
  // memcpy to device;
  cuda_status = hipMemcpy( dev_mat_i_lst, mat_i_lst, ( mat_idx * sizeof( mat_i_lst[0] ) ), hipMemcpyHostToDevice );
  if (cuda_status != hipSuccess)
    std::cout << "Device memcpy failed: dev_mat_i_lst" << std::endl;
  cuda_status = hipMemcpy( dev_v, v, ( vlen * sizeof( v[0] ) ), hipMemcpyHostToDevice );
  if (cuda_status != hipSuccess)
    std::cout << "Device memcpy failed: dev_v" << std::endl;
  cuda_status = hipMemcpy( dev_w, v, ( vlen * sizeof( v[0] ) ), hipMemcpyHostToDevice );
  if (cuda_status != hipSuccess)
    std::cout << "Device memcpy failed: dev_w" << std::endl;
  cuda_status = hipMemcpy( dev_w_med, v, ( vlen * sizeof( v[0] ) ), hipMemcpyHostToDevice );
  if (cuda_status != hipSuccess)
    std::cout << "Device memcpy failed: dev_w_med" << std::endl;
  cuda_status = hipMemcpy( dev_coeff_lst_zplx, coeff_lst_zplx, ( nTerm * sizeof( coeff_lst_zplx[0] ) ), hipMemcpyHostToDevice );
  if (cuda_status != hipSuccess)
    std::cout << "Device memcpy failed: dev_coeff_lst_zplx" << std::endl;
  
  hipblasCreate( &cublas_handle );
  
  hamvec_cuda3( cublas_handle, nspin, nTerm, coeff_lst_zplx, nbody_lst, pos_i_idx, pos_i_lst, dim_i_lst, mat_i_idx, dev_mat_i_lst, vlen, nspin_dim, nspin_m_lst, nspin_n_lst, dev_v, dev_w, dev_w_med, dev_coeff_lst_zplx, maxThreadsPerBlock, maxGridSize );
  
  cuda_status = hipMemcpy( w, dev_w, ( vlen * sizeof( w[0]) ), hipMemcpyDeviceToHost);
  if (cuda_status != hipSuccess)
    std::cout << "Device memcpy failed: w" << std::endl;
  
  hipblasDestroy( cublas_handle );
  hipFree( dev_mat_i_lst );
  hipFree( dev_v );
  hipFree( dev_w );
  hipFree( dev_w_med );
  hipFree( dev_coeff_lst_zplx );
  
}
